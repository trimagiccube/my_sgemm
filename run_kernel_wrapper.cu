#include <hipblas.h>
#include <stdexcept>
#include "kernels.cuh"


void runCublasFP32(hipblasHandle_t handle, int M, int N, int K, float alpha, float *A, float *B, float beta, float *C)
{
#if 0
	hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		N/*c row*/, M/*c column*/, K, &alpha,
		B/*matrix b*/, HIP_R_32F/*B is fp32*/, N/*B leading dimension*/,
		A, HIP_R_32F/*A is fp32*/, K/*A leading dimension*/,
		&beta, C, HIP_R_32F, N/*C leading dimension*/,
		HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
#endif
	hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		N, M, K, &alpha,
		B, HIP_R_32F, N,
		A, HIP_R_32F, K,
		&beta, C, HIP_R_32F, N,
		HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

void run_native(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C)
{
	dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
	dim3 blockDim(32, 32);
	native<<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void run_shared_memory_1(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C)
{
	dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
	dim3 blockDim(32 , 32);
#if 0
	hipFuncSetAttribute(reinterpret_cast<const void*>(shared_memory<32>),
			hipFuncAttributePreferredSharedMemoryCarveout,
			cudaSharedmemCarveoutMaxShared);
#endif
	shared_memory_1<32><<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void run_shared_memory_2(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C)
{
	dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
	dim3 blockDim(32 * 32);
#if 0
	hipFuncSetAttribute(reinterpret_cast<const void*>(shared_memory<32>),
			hipFuncAttributePreferredSharedMemoryCarveout,
			cudaSharedmemCarveoutMaxShared);
#endif
	shared_memory_2<32><<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void run_blocktile_1d_thread(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C)
{
	const uint BM = 64;
	const uint BN = 64;
	const uint BK = 8;
	const uint TM = 8;
	dim3 gridDim(CEIL_DIV(N, BN), CEIL_DIV(M, BM));
	dim3 blockDim((BM * BN) / TM);
	blocktile_1d_thread<BM, BN, BK, TM><<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void run_blocktile_2d_thread(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C)
{
	const uint BM = 128;
	const uint BN = 128;
	const uint BK = 8;
	const uint TM = 8;
	const uint TN = 8;
	dim3 gridDim(CEIL_DIV(N, BN), CEIL_DIV(M, BM));
	dim3 blockDim((BM * BN) / (TM * TN));
	blocktile_2d_thread<BM, BN, BK, TM, TN><<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void run_native_global_coalesce(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C)
{
	dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
	dim3 blockDim(32, 32);
	native_global_coalesce<<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void run_kernel(int kernel_num, hipblasHandle_t handle, int M, int N, int K, float alpha, float *A, float *B, float beta, float *C)
{
	switch (kernel_num) {
		case 0:
			runCublasFP32(handle, M, N, K, alpha, A, B, beta, C);
			break;
		case 1:
			run_native(M, N, K, alpha, A, B, beta, C);
			break;
		case 2:
			run_native_global_coalesce(M, N, K, alpha, A, B, beta, C);
			break;
		case 3:
			run_shared_memory_1(M, N, K, alpha, A, B, beta, C);
			break;
		case 4:
			run_shared_memory_2(M, N, K, alpha, A, B, beta, C);
			break;
		case 5:
			run_blocktile_1d_thread(M, N, K, alpha, A, B, beta, C);
			break;
		case 6:
			run_blocktile_2d_thread(M, N, K, alpha, A, B, beta, C);
			break;

		default:
			throw std::invalid_argument("invalid kernel number");

	}

}
