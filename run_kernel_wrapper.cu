#include <hipblas.h>
#include "kernels.cuh"


void runCublasFP32(hipblasHandle_t handle, int M, int N, int K, float alpha, float *A, float *B, float beta, float *C)
{
#if 0
	hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		N/*c row*/, M/*c column*/, K, &alpha,
		B/*matrix b*/, HIP_R_32F/*B is fp32*/, N/*B leading dimension*/,
		A, HIP_R_32F/*A is fp32*/, K/*A leading dimension*/,
		&beta, C, HIP_R_32F, N/*C leading dimension*/,
		HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
#endif
	hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		N, M, K, &alpha,
		B, HIP_R_32F, N,
		A, HIP_R_32F, K,
		&beta, C, HIP_R_32F, N,
		HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}
